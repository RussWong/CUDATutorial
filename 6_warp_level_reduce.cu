//warp shuffle
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#define WarpSize 32
//latency: 1.254ms
template <int blockSize>
__device__ float WarpShuffle(float sum) {
    // __shfl_down_sync：前面的thread向后面的thread要数据
    // __shfl_up_sync: 后面的thread向前面的thread要数据
    // 1. 返回前面的thread向后面的thread要的数据，比如__shfl_down_sync(0xffffffff, sum, 16)那就是返回16号线程，17号线程的数据
    // 2. 使用warp shuffle指令的数据交换不会出现warp在shared memory上交换数据时的不一致现象，这一点是由GPU driver完成，故无需任何sync, 比如syncwarp
    // 3. 15-19行的5个if存在的必要性: block Size为人为指定，那么有可能位于以下5个if的区间，所以需要这些if根据实际分配的block size来过滤操作
    if (blockSize >= 32)sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (blockSize >= 16)sum += __shfl_down_sync(0xffffffff, sum, 8);// 0-8, 1-9, 2-10, etc.
    if (blockSize >= 8)sum += __shfl_down_sync(0xffffffff, sum, 4);// 0-4, 1-5, 2-6, etc.
    if (blockSize >= 4)sum += __shfl_down_sync(0xffffffff, sum, 2);// 0-2, 1-3, 4-6, 5-7, etc.
    if (blockSize >= 2)sum += __shfl_down_sync(0xffffffff, sum, 1);// 0-1, 2-3, 4-5, etc.
    return sum;
}

template <int blockSize>
__global__ void reduce_warp_level(float *d_in,float *d_out, unsigned int n){
    float sum = 0;//当前线程的私有寄存器，即每个线程都会拥有一个sum寄存器

    unsigned int tid = threadIdx.x;
    unsigned int gtid = blockIdx.x * blockSize + threadIdx.x;
    // 分配的线程总数
    unsigned int total_thread_num = blockSize * gridDim.x;
    // 基于v5的改进：不用显式指定一个线程处理2个元素，而是通过L30的for循环来自动确定每个线程处理的元素个数
    for (int i = gtid; i < n; i += total_thread_num)
    {
        sum += d_in[i];
    }
    
    // 用于存储partial sums for each warp of a block
    __shared__ float WarpSums[blockSize / WarpSize]; 
    // 当前线程在其所在warp内的ID
    const int laneId = tid % WarpSize;
    // 当前线程所在warp在所有warp范围内的ID
    const int warpId = tid / WarpSize; 
    // 对当前线程所在warp作warpshuffle操作，直接交换warp内线程间的寄存器数据
    sum = WarpShuffle<blockSize>(sum);
    if(laneId == 0) {
        WarpSums[warpId] = sum;
    }
    __syncthreads();
    //至此，得到了每个warp的reduce sum结果
    //接下来，再使用第一个warp(laneId=0-31)对每个warp的reduce sum结果求和
    //首先，把warpsums存入前blockDim.x / WarpSize个线程的sum寄存器中
    //接着，继续warpshuffle
    sum = (tid < blockSize / WarpSize) ? WarpSums[laneId] : 0;
    // Final reduce using first warp
    if (warpId == 0) {
        sum = WarpShuffle<blockSize/WarpSize>(sum); 
    }
    // store: 哪里来回哪里去，把reduce结果写回显存
    if (tid == 0) {
        d_out[blockIdx.x] = sum;
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    //int GridSize = 100000;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float*)malloc((GridSize) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_warp_level<blockSize><<<Grid,Block>>>(d_a, d_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d \n", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < GridSize;i++){
            printf("resPerBlock : %lf ",out[i]);
        }
        printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_warp_level latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}
